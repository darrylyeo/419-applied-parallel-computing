#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

#define THREADS_PER_BLOCK 3

__global__ void calculate(char *buffer, double (*f) (double)){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < div){
		double x = start + i * step;
		buffer[i] = f(x);
	}
}

double integrate(double start, double end, int div, double (*f) (double)){
	double step = (end - start) / div;
	double result = (f(start) + f(end)) / 2;

	char *buffer;
	hipMallocManaged(&buffer, sizeof(int) * div);
	calculate<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(buffer, f);

	for(int i = 0; i < div; i++)
		result += buffer[i]; // f(start + i * step);
}

double f(double x){
	return x;
}

int main(void){
	hipError_t err = hipSuccess;

	integrate(st, en, div, f);
	
	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();

	for(int i = 0; i < N; i++)
		printf("%d ", buffer[i]);
	printf("\n");
	
	hipFree(buffer);
	
	return 0;
}