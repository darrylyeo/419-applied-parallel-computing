#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <helper_cuda.h>

#define THREADS_PER_BLOCK 3

double f(double x){
	return x*x;
}

__global__ void calculate(double *buffer, double start, double step, int N, double (*f) (double)){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < N){
		double x = start + i * step;
		buffer[i] = f(x);
	}
}

double integrate(double *buffer, double start, double end, int div, double (*f) (double)){
	int N = div;
	double step = (end - start) / div;

	hipMallocManaged(&buffer, sizeof(double) * N);
	calculate<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(buffer, start, step, N, f);

	hipDeviceSynchronize();

	for(int i = 0; i < N; i++)
		printf("%d ", buffer[i]);
	printf("\n");

	double result = (f(start) + f(end)) / 2;
	for(int i = 0; i < N; i++)
		result += buffer[i]; // f(start + i * step);
	return result;
}

int main(void){
	hipError_t err = hipSuccess;

	double *buffer = NULL;

	double result = integrate(buffer, 0, 10, 100, f);

	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Result: %d\n", result);

	hipFree(buffer);

	return 0;
}