#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

#define THREADS_PER_BLOCK 3

__global__ void calculate(double start, double step, int div, double (*f) (double), char *buffer){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if((double) i < div){
		double x = start + i * step;
		buffer[i] = f(x);
	}
}

double integrate(double start, double end, int div, double (*f) (double)){
	int N = div;
	double step = (end - start) / div;

	char *buffer;
	hipMallocManaged(&buffer, sizeof(int) * N);
	calculate<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(start, step, N, f, buffer);

	double result = (f(start) + f(end)) / 2;
	for(int i = 0; i < N; i++)
		result += buffer[i]; // f(start + i * step);
	return result;
}

double f(double x){
	return x*x;
}

int main(void){
	hipError_t err = hipSuccess;

	double result = integrate(0, 10, 0.1, f);
	
	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();

	printf("Result: %d\n", result);
	
	hipFree(buffer);
	
	return 0;
}