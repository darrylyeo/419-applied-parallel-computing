#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <helper_cuda.h>

__global__ void HelloWorld(){
	printf("Hello World");
}

int main(void){
	hipError_t err = hipSuccess;

	HelloWorld<<<5,1>>>();
	HelloWorld<<<5,5>>>();
	
	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();
	
	return 0;
}