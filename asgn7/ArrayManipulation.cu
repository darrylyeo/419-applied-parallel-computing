#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

#define N 20

__global__ void double(int *a){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < N)
		a[i] *= 2;
}

int main(void){
	hipError_t err = hipSuccess;

	size_t size = N * sizeof(int);
	int *a;
	hipMallocManaged(&a, size); // Use `a` on the CPU and/or on any GPU in the accelerated system.

	for(int i = 0; i < N; i++)
		a[i] = i;

	double<<<2,10>>>(&a);

	for(int i = 0; i < N; i++)
		print("%d ", a[i]);
	print("\n");
	
	hipFree(a);
	
	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();
	
	return 0;
}