#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

#define N 1000000

__global__ void addVectors(int *a, int *b, int *c){
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += gridDim.x * blockDim.x)
		c[i] = a[i] + b[i];
}

int main(void){
	hipError_t err = hipSuccess;

	size_t size = N * sizeof(int);

	int *a, *b, *c;
	hipMallocManaged(&a, size);
	hipMallocManaged(&b, size);
	hipMallocManaged(&c, size);

	for(int i = 0; i < N; i++){
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	size_t threads_per_block = 256;
	size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;
	addVectors<<<number_of_blocks, threads_per_block>>>(a, b, c);
	
	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();

	for(int i = 0; i < N; i++)
		printf("%d ", c[i];
	printf("\n");

	printf("Done\n");
	
	hipFree(a);
	
	return 0;
}