#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <helper_cuda.h>

#define N 1024
#define THREADS_PER_BLOCK 32

__global__ void SingleBlockLoop(){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	printf("%d\n", i);
}

int main(void){
	hipError_t err = hipSuccess;

	SingleBlockLoop<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, N>>>();
	
	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();
	
	return 0;
}