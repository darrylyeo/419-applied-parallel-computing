#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <helper_cuda.h>

#define N 1024
#define THREADS_PER_BLOCK 32

__global__ void SingleBlockLoop(){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < N)
		printf("%d\n", i);
}

int main(void){
	hipError_t err = hipSuccess;

	SingleBlockLoop<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>();
	
	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();
	
	return 0;
}