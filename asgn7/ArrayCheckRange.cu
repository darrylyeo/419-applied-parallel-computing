#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

int N = 1000;
size_t threads_per_block = 256;
size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

__global__ void initializeElementsTo(int *a, int *value){
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < N)
		a[value] = *value;
}

int main(void){
	hipError_t err = hipSuccess;

	size_t size = N * sizeof(int);
	int *a;
	hipMallocManaged(&a, size);

	initializeElementsTo<<<number_of_blocks, threads_per_block>>>(a, 123);
	
	if ((err = hipGetLastError()) != hipSuccess){
		fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();

	for(int i = 0; i < N; i++)
		if(a[i] != 123){
			printf("Failed\n");
			break;
		}
	printf("Done\n");
	
	hipFree(a);
	
	return 0;
}